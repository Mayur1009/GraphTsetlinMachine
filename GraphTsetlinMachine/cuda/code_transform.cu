#include "hip/hip_runtime.h"
extern "C" {

__global__ void transform(int *global_clause_node_output, int number_of_nodes, int *transformed_X) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // int number_of_node_chunks = (number_of_nodes - 1) / INT_SIZE + 1;

    for (int clause = index; clause < CLAUSES; clause += stride) {
        int clause_output = 0;
        for (int n = 0; n < number_of_nodes; n++) {
            int chunk_nr = n / INT_SIZE;
            int chunk_pos = n % INT_SIZE;

            if (global_clause_node_output[clause * NODE_CHUNKS + chunk_nr] & (1 << chunk_pos)) {
                clause_output = 1;
                break;
            }
        }
        if (clause_output)
            transformed_X[clause] = 1;
        else
            transformed_X[clause] = 0;
    }
}

__global__ void transform_nodewise(int *global_clause_node_output, int number_of_nodes, int *transformed_X) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // int number_of_node_chunks = (number_of_nodes - 1) / INT_SIZE + 1;
    for (int clause = index; clause < CLAUSES; clause += stride) {
        for (int n = 0; n < number_of_nodes; n++) {
            int chunk_nr = n / INT_SIZE;
            int chunk_pos = n % INT_SIZE;

            transformed_X[clause * number_of_nodes + n] =
                (global_clause_node_output[clause * NODE_CHUNKS + chunk_nr] & (1 << chunk_pos)) > 0;
        }
    }
}
}
