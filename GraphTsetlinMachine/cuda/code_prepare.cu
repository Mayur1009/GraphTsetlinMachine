#include "hip/hip_runtime.h"
extern "C" {
__global__ void prepare_message_ta_state(unsigned int *global_ta_state) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (unsigned long long clause = index; clause < CLAUSES; clause += stride) {
        unsigned int *ta_state = &global_ta_state[clause * MESSAGE_CHUNKS * STATE_BITS];
        for (int message_ta_chunk = 0; message_ta_chunk < MESSAGE_CHUNKS; ++message_ta_chunk) {
            for (int b = 0; b < STATE_BITS - 1; ++b) {
                ta_state[message_ta_chunk * STATE_BITS + b] = ~0;
            }
            ta_state[message_ta_chunk * STATE_BITS + STATE_BITS - 1] = 0;
        }
    }
}

__global__ void prepare(hiprandState *state, unsigned int *global_ta_state, int *clause_weights, int *class_sum) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState localState = state[index];

    for (unsigned long long clause = index; clause < CLAUSES; clause += stride) {
        for (unsigned long long class_id = 0; class_id < CLASSES; ++class_id) {
#if NEGATIVE_CLAUSES == 1
            clause_weights[class_id * CLAUSES + clause] =
                1 - 2 * (hiprand(&localState) % 2);  // 1 - 2*(clause % CLASSES != class_id);
#else
            clause_weights[class_id * CLAUSES + clause] = 1;
#endif
        }

        unsigned int *ta_state = &global_ta_state[clause * LA_CHUNKS * STATE_BITS];
        for (int la_chunk = 0; la_chunk < LA_CHUNKS; ++la_chunk) {
            for (int b = 0; b < STATE_BITS - 1; ++b) {
                ta_state[la_chunk * STATE_BITS + b] = ~0;
            }
            ta_state[la_chunk * STATE_BITS + STATE_BITS - 1] = 0;
        }
    }

    state[index] = localState;
}
}
