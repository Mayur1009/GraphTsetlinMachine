#include "hip/hip_runtime.h"
extern "C" {

// Increment the states of each of those 32 Tsetlin Automata flagged in the active bit vector.
__device__ inline void inc(unsigned int *ta_state, int chunk, unsigned int active) {
    unsigned int carry, carry_next;
    int id = chunk * STATE_BITS;
    carry = active;
    for (int b = 0; b < STATE_BITS; ++b) {
        if (carry == 0) break;

        carry_next = ta_state[id + b] & carry;        // Sets carry bits (overflow) passing on to next bit
        ta_state[id + b] = ta_state[id + b] ^ carry;  // Performs increments with XOR
        carry = carry_next;
    }

    if (carry > 0) {
        for (int b = 0; b < STATE_BITS; ++b) {
            ta_state[id + b] |= carry;
        }
    }
}

// Decrement the states of each of those 32 Tsetlin Automata flagged in the active bit vector.
__device__ inline void dec(unsigned int *ta_state, int chunk, unsigned int active) {
    unsigned int carry, carry_next;
    int id = chunk * STATE_BITS;
    carry = active;
    for (int b = 0; b < STATE_BITS; ++b) {
        if (carry == 0) break;

        carry_next = (~ta_state[id + b]) & carry;     // Sets carry bits (overflow) passing on to next bit
        ta_state[id + b] = ta_state[id + b] ^ carry;  // Performs increments with XOR
        carry = carry_next;
    }

    if (carry > 0) {
        for (int b = 0; b < STATE_BITS; ++b) {
            ta_state[id + b] &= ~carry;
        }
    }
}

__device__ inline void update_clause_message(hiprandState *localState, float s, int target_sign, unsigned int *ta_state,
                                             int clause_output, int clause_node, int number_of_include_actions,
                                             int *X) {
    if (target_sign > 0) {
        // Type I Feedback
        for (int la_chunk = 0; la_chunk < MESSAGE_CHUNKS; ++la_chunk) {
            // Generate random bit values
            unsigned int la_feedback = 0;
            for (int b = 0; b < INT_SIZE; ++b) {
                if (hiprand_uniform(localState) <= 1.0 / s) {
                    la_feedback |= (1 << b);
                }
            }

            if (clause_output && number_of_include_actions <= MAX_INCLUDED_LITERALS) {
#if BOOST_TRUE_POSITIVE_FEEDBACK == 1
                inc(ta_state, la_chunk, X[clause_node * MESSAGE_CHUNKS + la_chunk]);
#else
                inc(ta_state, la_chunk, X[clause_node * MESSAGE_CHUNKS + la_chunk] & (~la_feedback));
#endif

                dec(ta_state, la_chunk, (~X[clause_node * MESSAGE_CHUNKS + la_chunk]) & la_feedback);
            } else {
                dec(ta_state, la_chunk, la_feedback);
            }
        }
    } else if (target_sign < 0 && clause_output) {
        // Type II Feedback

        for (int la_chunk = 0; la_chunk < MESSAGE_CHUNKS; ++la_chunk) {
            inc(ta_state, la_chunk,
                (~X[clause_node * MESSAGE_CHUNKS + la_chunk]) & (~ta_state[la_chunk * STATE_BITS + STATE_BITS - 1]));
        }
    }
}

__device__ inline void update_clause(hiprandState *localState, float s, int target_sign, unsigned int *ta_state,
                                     int clause_output, int clause_node, int number_of_include_actions, int *X) {
    if (target_sign > 0) {
        // Type I Feedback
        for (int la_chunk = 0; la_chunk < LA_CHUNKS; ++la_chunk) {
            // Generate random bit values
            unsigned int la_feedback = 0;
            for (int b = 0; b < INT_SIZE; ++b) {
                if (hiprand_uniform(localState) <= 1.0 / s) {
                    la_feedback |= (1 << b);
                }
            }

            if (clause_output && number_of_include_actions <= MAX_INCLUDED_LITERALS) {
#if BOOST_TRUE_POSITIVE_FEEDBACK == 1
                inc(ta_state, la_chunk, X[clause_node * LA_CHUNKS + la_chunk]);
#else
                inc(ta_state, la_chunk, X[clause_node * LA_CHUNKS + la_chunk] & (~la_feedback));
#endif

                dec(ta_state, la_chunk, (~X[clause_node * LA_CHUNKS + la_chunk]) & la_feedback);
            } else {
                dec(ta_state, la_chunk, la_feedback);
            }
        }
    } else if (target_sign < 0 && clause_output) {
        // Type II Feedback

        for (int la_chunk = 0; la_chunk < LA_CHUNKS; ++la_chunk) {
            inc(ta_state, la_chunk,
                (~X[clause_node * LA_CHUNKS + la_chunk]) & (~ta_state[la_chunk * STATE_BITS + STATE_BITS - 1]));
        }
    }
}

__global__ void update_message(hiprandState *state, float s, unsigned int *global_ta_state, int number_of_nodes,
                               int *clause_node, int *number_of_include_actions, int *X, int *class_clause_update) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState localState = state[index];

    // Calculate clause output first
    for (unsigned long long clause = index; clause < CLAUSES; clause += stride) {
        unsigned int *ta_state = &global_ta_state[clause * MESSAGE_CHUNKS * STATE_BITS];

        for (unsigned long long class_id = 0; class_id < CLASSES; ++class_id) {
            update_clause_message(&localState, s, class_clause_update[class_id * CLAUSES + clause], ta_state,
                                  clause_node[clause] != -1, clause_node[clause], number_of_include_actions[clause], X);
        }
    }

    state[index] = localState;
}

__global__ void update(hiprandState *state, float s, unsigned int *global_ta_state, int number_of_nodes, int graph_index,
                       int *clause_node, int *number_of_include_actions, int *X, int *class_clause_update) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState localState = state[index];

    X = &X[graph_index * LA_CHUNKS];

    // Calculate clause output first
    for (unsigned long long clause = index; clause < CLAUSES; clause += stride) {
        unsigned int *ta_state = &global_ta_state[clause * LA_CHUNKS * STATE_BITS];

        for (unsigned long long class_id = 0; class_id < CLASSES; ++class_id) {
            update_clause(&localState, s, class_clause_update[class_id * CLAUSES + clause], ta_state,
                          clause_node[clause] != -1, clause_node[clause], number_of_include_actions[clause], X);
        }
    }

    state[index] = localState;
}
}
