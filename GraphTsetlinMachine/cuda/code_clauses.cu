#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

extern "C" {

__global__ void get_ta_states(unsigned int *ta_state, unsigned int chunks, unsigned int literals, unsigned int *out) {
    // :param: global_ta_state
    // Array of TAs for each literal.
    //
    // :param: chunks
    // Number of chunks.
    //
    // :param: literals
    // Number of literals.
    //
    // Shape:
    // For Clauses:
    //        (         CLAUSES,        LA_CHUNKS,           STATE_BITS)
    //        (number of clauses, number of chunks, number of state bits)
    // For Messages:
    //        (         CLAUSES,            MESSAGE_CHUNKS,           STATE_BITS)
    //        (number of clauses, number of message chunks, number of state bits)
    //
    //
    // :param: out
    // Output array to store the state values of each TA.
    // Shape: (CLAUSES, literals)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int clause = index; clause < CLAUSES; clause += stride) {
        unsigned int *local_ta_state = &ta_state[clause * chunks * STATE_BITS];

        for (int literal = 0; literal < literals; ++literal) {
            unsigned int state = 0;
            int chunk_nr = literal / INT_SIZE;
            int chunk_pos = literal % INT_SIZE;

            for (int bit = 0; bit < STATE_BITS; ++bit)
                if (local_ta_state[chunk_nr * STATE_BITS + bit] & (1 << chunk_pos)) state |= (1 << bit);

            out[clause * literals + literal] = state;
        }
    }
}
}
